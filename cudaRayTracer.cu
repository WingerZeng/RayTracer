#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>  //ͷ�ļ�
#include <>
#include "cudaRayTracer.h"

//__global__ void test() {
//	printf("Hi Cuda World");
//}
//
//int main(int argc, char** argv)
//{
//	test << <1, 1 >> > ();
//	hipDeviceSynchronize();
//	return 0;
//}


//__global__ void cudaRayColor(Scene* scene, std::vector<Ray>* ray, double t0, double t1, int jumptime, std::vector<Color>* color) {
//	(*color)[threadIdx.x] = scene->rayColor((*ray)[threadIdx.x], t0, t1, jumptime);
//}
//
//void getRayColor(Scene* scene, std::vector<Ray> ray, double t0, double t1, int jumptime, std::vector<Color>& color){
//	cudaRayColor<<<1,ray.size()>>>(scene,&ray,t0,t1,jumptime,&color);
//	hipDeviceSynchronize();
//}
//
